#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_WIDTH 16

__global__ void matrixMulKernel(float *A, float *B, float *C, int width) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Cvalue = 0;

    for (int m = 0; m < width/TILE_WIDTH; ++m) {
        ds_A[ty][tx] = A[Row*width + (m*TILE_WIDTH + tx)];
        ds_B[ty][tx] = B[(m*TILE_WIDTH + ty)*width + Col];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            Cvalue += ds_A[ty][k] * ds_B[k][tx];
        }
        __syncthreads();
    }
    C[Row*width + Col] = Cvalue;
}

int main() {
    int width = 1024;
    size_t size = width * width * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < width * width; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid(width / TILE_WIDTH, width / TILE_WIDTH);
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("First few elements of the result matrix:\n");
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%f ", h_C[i * width + j]);
        }
        printf("\n");
    }

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
