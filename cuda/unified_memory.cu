#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void increment(int *a, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        a[i]++;
    }
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(int);

    int *a;
    hipMallocManaged(&a, size);

    for (int i = 0; i < N; i++) {
        a[i] = i;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    increment<<<numBlocks, blockSize>>>(a, N);

    hipDeviceSynchronize();

    printf("First few elements after increment:\n");
    for (int i = 0; i < 10; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");

    hipFree(a);

    return 0;
}
