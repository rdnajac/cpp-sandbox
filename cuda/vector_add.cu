#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1000000;
    size_t size = n * sizeof(float);

    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    for (int i = 0; i < n; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }

    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
