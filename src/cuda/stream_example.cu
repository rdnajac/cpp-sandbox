#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000
#define STREAMS 4

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    hipStream_t streams[STREAMS];

    hipHostMalloc(&h_a, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_b, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_c, N * sizeof(float), hipHostMallocDefault);

    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        h_a[i] = rand() / (float)RAND_MAX;
        h_b[i] = rand() / (float)RAND_MAX;
    }

    for (int i = 0; i < STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    int streamSize = N / STREAMS;
    int blockSize = 256;
    int numBlocks = (streamSize + blockSize - 1) / blockSize;

    for (int i = 0; i < STREAMS; i++) {
        int offset = i * streamSize;
        hipMemcpyAsync(&d_a[offset], &h_a[offset], streamSize * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&d_b[offset], &h_b[offset], streamSize * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        vectorAdd<<<numBlocks, blockSize, 0, streams[i]>>>(&d_a[offset], &d_b[offset], &d_c[offset], streamSize);
        hipMemcpyAsync(&h_c[offset], &d_c[offset], streamSize * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();

    printf("First few elements of the result:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }

    for (int i = 0; i < STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
