#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256

__global__ void reduce(float *input, float *output, int n) {
    __shared__ float sdata[BLOCK_SIZE];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? input[i] : 0;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) output[blockIdx.x] = sdata[0];
}

int main() {
    int n = 1000000;
    size_t size = n * sizeof(float);

    float *h_input = (float *)malloc(size);
    for (int i = 0; i < n; i++) {
        h_input[i] = rand() / (float)RAND_MAX;
    }

    float *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, (n / BLOCK_SIZE) * sizeof(float));

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    reduce<<<numBlocks, BLOCK_SIZE>>>(d_input, d_output, n);

    float *h_output = (float *)malloc(numBlocks * sizeof(float));
    hipMemcpy(h_output, d_output, numBlocks * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0;
    for (int i = 0; i < numBlocks; i++) {
        sum += h_output[i];
    }

    printf("Sum: %f\n", sum);

    free(h_input); free(h_output);
    hipFree(d_input); hipFree(d_output);

    return 0;
}
